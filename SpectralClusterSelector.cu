#include "hip/hip_runtime.h"
#include "SpectralClusterSelector.h"
#include <device_atomic_functions.h>
#include <>
#include <stdio.h>

const int thread_per_dim = 32;
const int thread_per_block = thread_per_dim * thread_per_dim;
const int max_intensity = 4096;
const int max_intensity_square = max_intensity * max_intensity;

void compute_similar_matrix_gpu(unsigned short *h_img_data, double *h_joint_entropy,
                                int rows, int cols, int band_count) {
    dim3 thread_size(thread_per_dim, thread_per_dim);
    dim3 block_size;
    block_size.x = (cols + thread_size.x - 1) / thread_size.x;
    block_size.y = (rows + thread_size.y - 1) / thread_size.y;
    int entropy_block_size = (max_intensity_square + thread_per_block - 1) / thread_per_block;
    unsigned short *d_img_data;
    hipMalloc(&d_img_data, rows * cols * band_count * sizeof(unsigned short));
    hipMemcpy(d_img_data, h_img_data, rows * cols * band_count * sizeof(unsigned short), hipMemcpyHostToDevice);
    int *d_joint_histogram;
    hipMalloc(&d_joint_histogram, max_intensity * max_intensity * sizeof(int));
    double *d_joint_entropy_partial, entropy_sum;
    hipMalloc(&d_joint_entropy_partial, max_intensity_square * sizeof(double));

    for(int i = 0; i < band_count; ++i) {
        for (int j = i; j < band_count; ++j) {
            hipMemset(d_joint_histogram, 0, max_intensity * max_intensity * sizeof(int));
            compute_joint_histogram<<<block_size, thread_size>>>(d_img_data,
                                                                 d_joint_histogram,
                                                                 i, j,
                                                                 rows * cols, cols, rows,
                                                                 band_count);

            compute_joint_entropy<<<entropy_block_size, thread_per_block>>>(d_joint_histogram,
                                                                       d_joint_entropy_partial, rows * cols);

//            double* tmp_buf = new double[max_intensity_square];
//            hipMemcpy(tmp_buf, d_joint_entropy_partial, max_intensity_square *sizeof(double), hipMemcpyDeviceToHost);
//            double sum = 0;
//            for(int k = 0; k < max_intensity_square; ++k)
//            {
//                sum += tmp_buf[k];
//            }

            int d_joint_entropy_partial_len = max_intensity_square;
            int grid_dim = entropy_block_size;
            while (grid_dim > 0){
                sum_array<<<grid_dim, thread_per_block>>>(d_joint_entropy_partial,
                        d_joint_entropy_partial,
                        d_joint_entropy_partial_len);

                if (grid_dim == 1) {
                    break;
                }
                d_joint_entropy_partial_len = grid_dim;
                grid_dim = (grid_dim + thread_per_block - 1) / thread_per_block;
            };

            hipMemcpy(&entropy_sum, d_joint_entropy_partial, sizeof(double), hipMemcpyDeviceToHost);

            h_joint_entropy[i * band_count + j] = h_joint_entropy[j * band_count + i] = entropy_sum;

        }
    }

    hipFree(d_joint_entropy_partial);
    hipFree(d_joint_histogram);
    hipFree(d_img_data);
}

__global__ void compute_joint_histogram(unsigned short *d_img_data, int *d_joint_histogram,
                                        int band_idx1, int band_idx2, int pixel_count, int cols, int rows, int band_count) {
    int pixel_idx_x = blockIdx.x * blockDim.x + threadIdx.x;
    int pixel_idx_y = blockIdx.y * blockDim.y + threadIdx.y;

    if (pixel_idx_x < cols && pixel_idx_y < rows) {
        unsigned short pixel_idx1 = d_img_data[band_idx1 * pixel_count + pixel_idx_y * cols + pixel_idx_x];
        unsigned short pixel_idx2 = d_img_data[band_idx2 * pixel_count + pixel_idx_y * cols + pixel_idx_x];
        atomicAdd(d_joint_histogram + pixel_idx1 * band_count + pixel_idx2, 1);
    }

}

__global__ void compute_joint_entropy(int* d_joint_histogram, double *d_joint_entropy_partial, int pixel_count) {
    int his_idx = blockIdx.x * thread_per_block + threadIdx.x;

    if (his_idx < max_intensity_square && 0 != d_joint_histogram[his_idx]) {
        double probability = d_joint_histogram[his_idx] / (double)pixel_count;
        d_joint_entropy_partial[his_idx] = -probability * log2(probability);
    } else {
        d_joint_entropy_partial[his_idx] = 0.0;
    }
}

__global__ void sum_array(double* d_array_in, double* d_array_out, int len) {
    __shared__ double partial_sum[thread_per_block];
    int tid = threadIdx.x;
    int array_idx = blockIdx.x * blockDim.x + tid;

    if (array_idx > len) {
       partial_sum[tid] = 0.0;
    } else {
        partial_sum[tid] = d_array_in[array_idx];
    }
    __syncthreads();

    for(int stride = thread_per_block / 2; stride > 0; stride/=2)
    {
        if(tid < stride) {
            partial_sum[tid]+= partial_sum[tid + stride];
        }
        __syncthreads();
    }

    if (tid == 0) {
        d_array_out[blockIdx.x] = partial_sum[0];
    }

}